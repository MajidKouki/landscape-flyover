
#include <hip/hip_runtime.h>

// The code blocks below are commented out as they aren't meant to run as is
// but be integrated into the program as it develops

// // Draw diagonal lines across mesh
// static _global_ void updateHeightmapKernel(float *heightMap,
//                             unsigned int width, int timestep) {
//     unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//     unsigned int i = y * width + x;

//     heightMap[i] = (float)(((x + y + timestep) * 2) % 256) / 256.0f;
// }



// // Given x and y, copy a pixel from one map to another
// float pixel;
// if (y < shift) {
//     unsigned int row = (mapHeight - shift) + y;
//     pixel = *(map1 + x + (row * mapWidth));
// } else {
//     unsigned int row = y - shift;
//     pixel = *(map0 + x + (row * mapWidth));
// }



// // Viewpoint control using CPU
// if (autoFollow && (bottomRow != nullptr)) {
//     float y = bottomRow[meshWidth / 2];
//     targetTransY = anim_base_y - (y / 5.0f);
// }



// // Allocate GPU memory
// cudaMalloc(&heightmap, numPoints * sizeof(float));



// // Allocate CPU memroy
// cudaMallocHost(&bottomRow, meshWidth * sizeof(float));



// // Free allocated memory
// cudaFree(heightmap);
// cudaFreeHost(bottomRow);
// return 0;